#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void test_print_kernel(const float* pdata, int ndata){  // __global__:由host调用的核函数 ，需返回void 

    int idx = threadIdx.x;  // threadIdx, ...内置变量, gridDim(1,1,1), blockDim(>1,1,1)
    // int idx = threadIdx.x + blockIdx.x * blockDim.x;  //  gridDim(>1,1,1), blockDim(>1,1,1)时候的公式
    /*    dims                 indexs
        gridDim.z            blockIdx.z
        gridDim.y            blockIdx.y
        gridDim.x            blockIdx.x
        blockDim.z           threadIdx.z
        blockDim.y           threadIdx.y
        blockDim.x           threadIdx.x

        Pseudo code:
        position = 0
        for i in 6:
            position *= dims[i]
            position += indexs[i]
    */
    printf("Element[%d] = %f, threadIdx.x=%d, blockIdx.x=%d, blockDim.x=%d\n",
            idx, pdata[idx], threadIdx.x, blockIdx.x, blockDim.x);
}

void test_print(const float* pdata, int ndata){  // __host__调用

    // <<<gridDim, blockDim, bytes_of_shared_memory, stream>>>
    test_print_kernel<<<1, ndata, 0, nullptr>>>(pdata, ndata);  // 仅能传值，nullptr表示默认流

    // 在核函数执行结束后，通过cudaPeekAtLastError获取得到的代码，来知道是否出现错误
    // cudaPeekAtLastError和cudaGetLastError都可以获取得到错误代码
    // cudaGetLastError是获取错误代码并清除掉，也就是再一次执行cudaGetLastError获取的会是success
    // 而cudaPeekAtLastError是获取当前错误，但是再一次执行 cudaPeekAtLastError 或者 cudaGetLastError 拿到的还是那个错
    // cuda的错误会传递，如果这里出错了，不移除。那么后续的任意api的返回值都会是这个错误，都会失败
    hipError_t code = hipPeekAtLastError();
    if(code != hipSuccess){    
        const char* err_name    = hipGetErrorName(code);    
        const char* err_message = hipGetErrorString(code);  
        printf("kernel error %s:%d  test_print_kernel failed. \n  code = %s, message = %s\n",
             __FILE__, __LINE__, err_name, err_message);   
    }
}